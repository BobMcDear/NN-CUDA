#include "module.h"
#include "utils.h"
#include "../CPU/linear_cpu.h"
#include "../CPU/mse_cpu.h"
#include "../CPU/relu_cpu.h"
#include "../CPU/sequential_cpu.h"
#include "../GPU/linear_gpu.h"
#include "../GPU/mse_gpu.h"
#include "../GPU/relu_gpu.h"
#include "../GPU/sequential_gpu.h"

#include <chrono>
#include <iostream>
using namespace std;


int main(){
    /*
    int bs = 8, n_in = 9, n_hidden = 10, n_out = 1;
    int sz_inp = bs*n_in, sz_hidden = bs*n_hidden, sz_out = bs*n_out, sz_weight1 = n_in*n_hidden, sz_weight2 = n_hidden*n_out;

    float *inp = new float[sz_inp], *out = new float[sz_out], *targ = new float[sz_out];
    
    fill_array(inp, sz_inp, 1);
    fill_array(targ, sz_out, 2);

    cout << "inp:" << endl;
    //print_array(inp, sz_inp);

    cout << "targ:" << endl;
    //print_array(targ, sz_out);

    Linear_CPU* lin1 = new Linear_CPU(bs, n_in, n_hidden);
    ReLU_CPU* relu1 = new ReLU_CPU(sz_hidden);
    Linear_CPU* lin2 = new Linear_CPU(bs, n_hidden, n_out);
    ReLU_CPU* relu2 = new ReLU_CPU(sz_out);


    std::vector<Module*> layers = {lin1, relu1, lin2, relu2};
    Sequential_CPU seq(layers);
    MSE_CPU mse(sz_out);

    cout << "lin1->weights:" << endl;
    //print_array(lin1->weights, sz_weight1);

    cout << "lin2->weights:" << endl;
    //print_array(lin2->weights, sz_weight2);

    seq.forward(inp, out);
    mse.forward(relu2->out, targ);

    mse.backward();
    seq.update();

    cout << "Updated lin2->weights:" << endl;
    //print_array(lin2->weights, n_hidden*n_out);

    cout << "Updated lin2->bias:" << endl;
    //print_array(lin2->bias, n_out);
    
    cout << "Updated lin1->weights:" << endl;
    //print_array(lin1->weights, n_in*n_hidden);

    cout << "Updated lin1->bias:" << endl;
    //print_array(lin1->bias, n_hidden);
    */


    int bs = 128, n_in = 768, n_hidden1 = 100, n_hidden2 = 50, n_out = 1;
    int sz_inp = bs*n_in, sz_weight1 = n_in*n_hidden1, sz_weight2 = n_hidden1*n_hidden2, sz_weight3 = n_hidden2*n_out, sz_out = bs*n_out;

    float *inp_cpu = new float[sz_inp], *out_cpu = new float[sz_out], *targ_cpu = new float[sz_out];
    float *inp_gpu, *out_gpu, *targ_gpu;
    
    hipMallocManaged(&inp_gpu, sz_inp*sizeof(float));
    hipMallocManaged(&out_gpu, sz_out*sizeof(float));
    hipMallocManaged(&targ_gpu, sz_out*sizeof(float));

    fill_array(inp_cpu, sz_inp);
    fill_array(targ_cpu, sz_out);

    set_eq(inp_gpu, inp_cpu, sz_inp);
    set_eq(targ_gpu, targ_cpu, sz_out);

    //print_array(inp_cpu, sz_inp);
    //print_array(targ_cpu, sz_out);

    Linear_CPU* lin1_cpu = new Linear_CPU(bs, n_in, n_hidden1);
    ReLU_CPU* relu1_cpu = new ReLU_CPU(bs*n_hidden1);
    Linear_CPU* lin2_cpu = new Linear_CPU(bs, n_hidden1, n_hidden2);
    ReLU_CPU* relu2_cpu = new ReLU_CPU(bs*n_hidden2);
    Linear_CPU* lin3_cpu = new Linear_CPU(bs, n_hidden2, n_out);

    Linear_GPU* lin1_gpu = new Linear_GPU(bs, n_in, n_hidden1);
    ReLU_GPU* relu1_gpu = new ReLU_GPU(bs*n_hidden1);
    Linear_GPU* lin2_gpu = new Linear_GPU(bs, n_hidden1, n_hidden2);
    ReLU_GPU* relu2_gpu = new ReLU_GPU(bs*n_hidden2);
    Linear_GPU* lin3_gpu = new Linear_GPU(bs, n_hidden2, n_out);


    set_eq(lin1_gpu->weights, lin1_cpu->weights, sz_weight1);
    set_eq(lin2_gpu->weights, lin2_cpu->weights, sz_weight2);
    set_eq(lin3_gpu->weights, lin3_cpu->weights, sz_weight3);

    //print_array(lin1_cpu->weights, sz_weight1);
    //print_array(lin2_cpu->weights, sz_weight2);
    //print_array(lin3_cpu->weights, sz_weight3);


    std::vector<Module*> layers_cpu = {lin1_cpu, relu1_cpu, lin2_cpu, relu2_cpu, lin3_cpu};
    Sequential_CPU seq_cpu(layers_cpu);

    MSE_CPU mse_cpu(sz_out);

    chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
    for (int i=0; i<32; i++){
        seq_cpu.forward(inp_cpu, out_cpu);
        mse_cpu.forward(lin3_cpu->out, targ_cpu);

        mse_cpu.backward();
        seq_cpu.update();
    }
    std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();

    cout << "CPU time difference = " << chrono::duration_cast<chrono::microseconds>(end - begin).count() << "[µs]" << endl;

    
    std::vector<Module*> layers_gpu = {lin1_gpu, relu1_gpu, lin2_gpu, relu2_gpu, lin3_gpu};
    Sequential_GPU seq_gpu(layers_gpu);

    MSE_GPU mse_gpu(sz_out);

    chrono::steady_clock::time_point begin1 = std::chrono::steady_clock::now();
    for (int i=0; i<32; i++){
        seq_gpu.forward(inp_gpu, out_gpu);
        mse_gpu.forward(lin3_gpu->out, targ_gpu);

        mse_gpu.backward();
        seq_gpu.update();
    }
    std::chrono::steady_clock::time_point end1 = std::chrono::steady_clock::now();

    cout << "GPU time difference = " << chrono::duration_cast<chrono::milliseconds>(end1 - begin1).count() << "[µs]" << endl;

    //print_array(lin1_cpu->weights, sz_weight1);
    //print_array(lin2_cpu->weights, sz_weight2);
    //print_array(lin3_cpu->weights, sz_weight3);

    //test_res(lin1_gpu->weights, lin1_cpu->weights, sz_weight1);
    //test_res(lin2_gpu->weights, lin2_cpu->weights, sz_weight2);
    //test_res(lin3_gpu->weights, lin3_cpu->weights, sz_weight3);
    
    //test_res(lin1_gpu->bias, lin1_cpu->bias, n_hidden1);
    //test_res(lin2_gpu->bias, lin2_cpu->bias, n_hidden2);
    //test_res(lin3_gpu->bias, lin3_cpu->bias, n_out);

    return 0;
}
