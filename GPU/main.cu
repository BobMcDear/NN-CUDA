#include "linear.h"
#include "relu.h"
#include "train.h"
#include "../data/read_csv.h"

int main(){
    int bs = 10000, n_in = 10, n_epochs = 10;
    int n_hidden = n_in/2;

    float *inp, *targ;
    hipMallocManaged(&inp, bs*n_in*sizeof(float));
    hipMallocManaged(&targ, (bs+1)*sizeof(float));

    read_csv(inp, "../data/x.csv");
    read_csv(targ, "../data/y.csv");
    
    Linear_GPU* lin1 = new Linear_GPU(bs, n_in, n_hidden);
    ReLU_GPU* relu1 = new ReLU_GPU(bs*n_hidden);
    Linear_GPU* lin2 = new Linear_GPU(bs, n_hidden, 1);
    
    std::vector<Module*> layers = {lin1, relu1, lin2};
    Sequential_GPU seq(layers);
    
    train_gpu(seq, inp, targ, bs, n_in, n_epochs);

    return 0;
}
