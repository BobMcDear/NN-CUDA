#include "linear.h"
#include "relu.h"
#include "train.h"
#include "../data/read_csv.h"


int main(){
    int bs = 100000, n_in = 100, n_epochs = 10;
    int n_hidden1 = n_in/2, n_hidden2 = n_in/4, n_hidden3 = n_in/8;
    float *inp, *targ;

    hipMallocManaged(&inp, (bs*n_in)*sizeof(float));
    hipMallocManaged(&targ, (bs+1)*sizeof(float));

    read_csv(inp, "../data/x.csv");
    read_csv(targ, "../data/y.csv");
    
    Linear_GPU* lin1 = new Linear_GPU(bs, n_in, n_hidden1);
    ReLU_GPU* relu1 = new ReLU_GPU(bs*n_hidden1);
    Linear_GPU* lin2 = new Linear_GPU(bs, n_hidden1, n_hidden2);
    ReLU_GPU* relu2 = new ReLU_GPU(bs*n_hidden2);
    Linear_GPU* lin3 = new Linear_GPU(bs, n_hidden2, n_hidden3);
    ReLU_GPU* relu3 = new ReLU_GPU(bs*n_hidden3);
    Linear_GPU* lin4 = new Linear_GPU(bs, n_hidden3, 1);

    std::vector<Module*> layers = {lin1, relu1, lin2, relu2, lin3, relu3, lin4};
    Sequential_GPU seq(layers);
    
    train(seq, inp, targ, bs, n_in, n_epochs);

    return 0;
}
