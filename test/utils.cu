#include "hip/hip_runtime.h"
#include "utils.h"
#include "../utils/utils.h"


void get_data(float *inp_cpu, float *out_cpu, float *inp_gpu, float *out_gpu, int bs, int n_in, int n_out){
    int sz_inp = bs*n_in, sz_out = bs*n_out;

    inp_cpu = new float[sz_inp];
    out_cpu = new float[sz_out];

    hipMallocManaged(&inp_gpu, sz_inp*sizeof(float));
    hipMallocManaged(&out_gpu, sz_out*sizeof(float));

    fill_array(inp_cpu, sz_inp);
    set_eq(inp_gpu, inp_cpu, sz_inp);
}


int random_int(int min, int max){
    std::random_device rd;
    std::mt19937 rng(rd());
    std::uniform_int_distribution<int> dist(min, max);
    return dist(rng);
}
