#include <iostream>

#include "utils.h"
#include "../CPU/linear.h"
#include "../GPU/linear.h"
#include "../utils/utils.h"


int main(){
    int bs, n_in, n_out;
    int sz_inp, sz_weights, sz_out;
    float *inp_cpu, *out_cpu, *inp_gpu, *out_gpu;

    for (int i=0; i<10; i++){
        std::cout << "Iteration " << i << std::endl;

        bs = random_int(32, 256);
        n_in = random_int(32, 64);
        n_out = random_int(1, 32);
        
        sz_inp = bs*n_in;
        sz_weights = n_in*n_out;
        sz_out = bs*n_out;

        inp_cpu = new float[sz_inp];
        out_cpu = new float[sz_out];
    
        hipMallocManaged(&inp_gpu, sz_inp*sizeof(float));
        hipMallocManaged(&out_gpu, sz_out*sizeof(float));
    
        fill_array(inp_cpu, sz_inp);
        set_eq(inp_gpu, inp_cpu, sz_inp);
        
        Linear_CPU lin_cpu(bs, n_in, n_out);
        Linear_GPU lin_gpu(bs, n_in, n_out);
        set_eq(lin_gpu.weights, lin_cpu.weights, sz_weights);
    
        lin_cpu.forward(inp_cpu, out_cpu);
        lin_gpu.forward(inp_gpu, out_gpu);

        std::cout << "Result of the forward pass" << std::endl; 
        test_res(lin_cpu.out, lin_gpu.out, sz_out);    
    }
    
    return 0;
}
