#include <iostream>

#include "utils.h"
#include "../CPU/linear.h"
#include "../CPU/mse.h"
#include "../CPU/relu.h"
#include "../GPU/linear.h"
#include "../GPU/mse.h"
#include "../GPU/relu.h"
#include "../utils/utils.h"


int main(){
    int bs, n_in, n_out;
    int sz_inp, sz_weights;
    float *inp_cpu, *inp_gpu, *out;

    for (int i=0; i<10; i++){
        std::cout << "Iteration " << i << std::endl;
        
        bs = random_int(16, 256);
        n_in = random_int(32, 64);
        
        sz_inp = bs*n_in;
        sz_weights = n_in*n_out;

        inp_cpu = new float[sz_inp];
        hipMallocManaged(&inp_gpu, sz_inp*sizeof(float));
    
        fill_array(inp_cpu, sz_inp);
        set_eq(inp_gpu, inp_cpu, sz_inp);

        Linear_CPU* lin_cpu = new Linear_CPU(bs, n_in, n_out);
        Linear_GPU* lin_gpu = new Linear_GPU(bs, n_in, n_out);
        set_eq(lin_gpu.weights, lin_cpu.weights, sz_weights);

        ReLU_CPU* relu_cpu = new ReLU_CPU(bs);
        ReLU_GPU* relu_gpu = new ReLU_GPU(bs);

        std::vector<Module*> layers_cpu = {lin_cpu, relu_cpu};
        std::vector<Module*> layers_gpu = {lin_gpu, relu_gpu};

        Sequential_CPU seq_cpu(layers_cpu);
        Sequential_GPU seq_gpu(layers_gpu);

        seq_cpu.forward(inp_cpu, out);
        seq_gpu.forward(inp_gpu, out);

        std::cout << "Result of the forward pass" << std::endl; 
        test_res(layers_cpu.back()->out, layers_gpu.back()->out, sz_out);    
    }
    
    return 0;
}
