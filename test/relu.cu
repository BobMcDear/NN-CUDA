#include <iostream>

#include "utils.h"
#include "../CPU/relu.h"
#include "../GPU/relu.h"
#include "../utils/utils.h"


int main(){
    int bs;
    float *inp_cpu, *out_cpu, *inp_gpu, *out_gpu;

    for (int i=0; i<10; i++){
        std::cout << "Iteration " << i << std::endl;
        
        bs = random_int(128, 2048);

        inp_cpu = new float[bs];
        out_cpu = new float[bs];
    
        hipMallocManaged(&inp_gpu, bs*sizeof(float));
        hipMallocManaged(&out_gpu, bs*sizeof(float));
    
        fill_array(inp_cpu, bs);
        set_eq(inp_gpu, inp_cpu, bs);

        ReLU_CPU relu_cpu(bs);
        ReLU_GPU relu_gpu(bs);
    
        relu_cpu.forward(inp_cpu, out_cpu);
        relu_gpu.forward(inp_gpu, out_gpu);

        std::cout << "Result of the forward pass" << std::endl; 
        test_res(out_cpu, out_cpu, bs);
    }
    
    return 0;
}
