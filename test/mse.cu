#include <iostream>

#include "utils.h"
#include "../CPU/mse.h"
#include "../GPU/mse.h"
#include "../utils/utils.h"


int main(){
    int bs;
    float *inp_cpu, *out_cpu, *inp_gpu, *out_gpu;

    for (int i=0; i<10; i++){
        std::cout << "Iteration " << i << std::endl;
        
        bs = random_int(32, 2048);

        inp_cpu = new float[sz_inp];
        out_cpu = new float[sz_out];
    
        hipMallocManaged(&inp_gpu, bs*sizeof(float));
        hipMallocManaged(&out_gpu, bs*sizeof(float));
    
        fill_array(inp_cpu, bs);
        set_eq(inp_gpu, inp_cpu, bs);
        
        MSE_CPU mse_cpu(bs);
        MSE_GPU mse_gpu(bs);
    
        mse_cpu.forward(inp_cpu, out_cpu);
        mse_gpu.forward(inp_gpu, out_gpu);
        mse_cpu._forward(inp_cpu, out_cpu);
        mse_gpu._forward(inp_gpu, out_gpu);
    
        std::cout << "Result of the forward pass" << std::endl; 
        std::cout << out_cpu[bs]-out_gpu[bs] << std::endl;
        
        mse_cpu.backward();
        mse_gpu.backward();

        std::cout << "Result of the backward pass" << std::endl; 
        test_res(mse_cpu.inp, mse_gpu.inp, bs);
    }
    
    return 0;
}
